
#include <hip/hip_runtime.h>
/*!
 * Copyright (c) 2021 Microsoft Corporation. All rights reserved.
 * Licensed under the MIT License. See LICENSE file in the project root for
 * license information.
 * Modifications Copyright(C) 2023 Advanced Micro Devices, Inc. All rights reserved.
 */


#ifdef USE_CUDA

#include "cuda_leaf_splits.hpp"
#include <LightGBM/cuda/cuda_algorithms.hpp>
#include <LightGBM/cuda/cuda_rocm_interop.h>

namespace LightGBM {

template <bool USE_INDICES>
__global__ void CUDAInitValuesKernel1(const score_t* cuda_gradients, const score_t* cuda_hessians,
  const data_size_t num_data, const data_size_t* cuda_bagging_data_indices,
  double* cuda_sum_of_gradients, double* cuda_sum_of_hessians) {
  __shared__ double shared_mem_buffer[WARPSIZE];
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  double gradient = 0.0f;
  double hessian = 0.0f;
  if (data_index < num_data) {
    gradient = USE_INDICES ? cuda_gradients[cuda_bagging_data_indices[data_index]] : cuda_gradients[data_index];
    hessian = USE_INDICES ? cuda_hessians[cuda_bagging_data_indices[data_index]] : cuda_hessians[data_index];
  }
  const double block_sum_gradient = ShuffleReduceSum<double>(gradient, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double block_sum_hessian = ShuffleReduceSum<double>(hessian, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[blockIdx.x] += block_sum_gradient;
    cuda_sum_of_hessians[blockIdx.x] += block_sum_hessian;
  }
}

__global__ void CUDAInitValuesKernel2(
  const double lambda_l1,
  const double lambda_l2,
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  __shared__ double shared_mem_buffer[WARPSIZE];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_hessians[0] = sum_of_hessians;
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

template <bool USE_INDICES>
__global__ void CUDAInitValuesKernel3(const int16_t* cuda_gradients_and_hessians,
  const data_size_t num_data, const data_size_t* cuda_bagging_data_indices,
  double* cuda_sum_of_gradients, double* cuda_sum_of_hessians, int64_t* cuda_sum_of_hessians_hessians,
  const score_t* grad_scale_pointer, const score_t* hess_scale_pointer) {
  const score_t grad_scale = *grad_scale_pointer;
  const score_t hess_scale = *hess_scale_pointer;
  __shared__ int64_t shared_mem_buffer[WARPSIZE];
  const data_size_t data_index = static_cast<data_size_t>(threadIdx.x + blockIdx.x * blockDim.x);
  int64_t int_gradient = 0;
  int64_t int_hessian = 0;
  if (data_index < num_data) {
    int_gradient = USE_INDICES ? cuda_gradients_and_hessians[2 * cuda_bagging_data_indices[data_index] + 1] :
      cuda_gradients_and_hessians[2 * data_index + 1];
    int_hessian = USE_INDICES ? cuda_gradients_and_hessians[2 * cuda_bagging_data_indices[data_index]] :
      cuda_gradients_and_hessians[2 * data_index];
  }
  const int64_t block_sum_gradient = ShuffleReduceSum<int64_t>(int_gradient, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const int64_t block_sum_hessian = ShuffleReduceSum<int64_t>(int_hessian, shared_mem_buffer, blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_gradients[blockIdx.x] = block_sum_gradient * grad_scale;
    cuda_sum_of_hessians[blockIdx.x] = block_sum_hessian * hess_scale;
    cuda_sum_of_hessians_hessians[blockIdx.x] = ((block_sum_gradient << 32) | block_sum_hessian);
  }
}

__global__ void CUDAInitValuesKernel4(
  const double lambda_l1,
  const double lambda_l2,
  const int num_blocks_to_reduce,
  double* cuda_sum_of_gradients,
  double* cuda_sum_of_hessians,
  int64_t* cuda_sum_of_gradients_hessians,
  const data_size_t num_data,
  const data_size_t* cuda_data_indices_in_leaf,
  hist_t* cuda_hist_in_leaf,
  CUDALeafSplitsStruct* cuda_struct) {
  __shared__ double shared_mem_buffer[WARPSIZE];
  double thread_sum_of_gradients = 0.0f;
  double thread_sum_of_hessians = 0.0f;
  int64_t thread_sum_of_gradients_hessians = 0;
  for (int block_index = static_cast<int>(threadIdx.x); block_index < num_blocks_to_reduce; block_index += static_cast<int>(blockDim.x)) {
    thread_sum_of_gradients += cuda_sum_of_gradients[block_index];
    thread_sum_of_hessians += cuda_sum_of_hessians[block_index];
    thread_sum_of_gradients_hessians += cuda_sum_of_gradients_hessians[block_index];
  }
  const double sum_of_gradients = ShuffleReduceSum<double>(thread_sum_of_gradients, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_hessians = ShuffleReduceSum<double>(thread_sum_of_hessians, shared_mem_buffer, blockDim.x);
  __syncthreads();
  const double sum_of_gradients_hessians = ShuffleReduceSum<int64_t>(
    thread_sum_of_gradients_hessians,
    reinterpret_cast<int64_t*>(shared_mem_buffer),
    blockDim.x);
  if (threadIdx.x == 0) {
    cuda_sum_of_hessians[0] = sum_of_hessians;
    cuda_struct->leaf_index = 0;
    cuda_struct->sum_of_gradients = sum_of_gradients;
    cuda_struct->sum_of_hessians = sum_of_hessians;
    cuda_struct->sum_of_gradients_hessians = sum_of_gradients_hessians;
    cuda_struct->num_data_in_leaf = num_data;
    const bool use_l1 = lambda_l1 > 0.0f;
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->gain = CUDALeafSplits::GetLeafGain<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    if (!use_l1) {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<false, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    } else {
      // no smoothing on root node
      cuda_struct->leaf_value =
        CUDALeafSplits::CalculateSplittedLeafOutput<true, false>(sum_of_gradients, sum_of_hessians, lambda_l1, lambda_l2, 0.0f, 0, 0.0f);
    }
    cuda_struct->data_indices_in_leaf = cuda_data_indices_in_leaf;
    cuda_struct->hist_in_leaf = cuda_hist_in_leaf;
  }
}

__global__ void InitValuesEmptyKernel(CUDALeafSplitsStruct* cuda_struct) {
  cuda_struct->leaf_index = -1;
  cuda_struct->sum_of_gradients = 0.0f;
  cuda_struct->sum_of_hessians = 0.0f;
  cuda_struct->num_data_in_leaf = 0;
  cuda_struct->gain = 0.0f;
  cuda_struct->leaf_value = 0.0f;
  cuda_struct->data_indices_in_leaf = nullptr;
  cuda_struct->hist_in_leaf = nullptr;
}

void CUDALeafSplits::LaunchInitValuesEmptyKernel() {
  InitValuesEmptyKernel<<<1, 1>>>(cuda_struct_.RawData());
}

void CUDALeafSplits::LaunchInitValuesKernel(
  const double lambda_l1, const double lambda_l2,
  const data_size_t* cuda_bagging_data_indices,
  const data_size_t* cuda_data_indices_in_leaf,
  const data_size_t num_used_indices,
  hist_t* cuda_hist_in_leaf) {
  if (cuda_bagging_data_indices == nullptr) {
    CUDAInitValuesKernel1<false><<<num_blocks_init_from_gradients_, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
      cuda_gradients_, cuda_hessians_, num_used_indices, nullptr, cuda_sum_of_gradients_buffer_.RawData(),
      cuda_sum_of_hessians_buffer_.RawData());
  } else {
    CUDAInitValuesKernel1<true><<<num_blocks_init_from_gradients_, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
      cuda_gradients_, cuda_hessians_, num_used_indices, cuda_bagging_data_indices, cuda_sum_of_gradients_buffer_.RawData(),
      cuda_sum_of_hessians_buffer_.RawData());
  }
  SynchronizeCUDADevice(__FILE__, __LINE__);
  CUDAInitValuesKernel2<<<1, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
    lambda_l1, lambda_l2,
    num_blocks_init_from_gradients_,
    cuda_sum_of_gradients_buffer_.RawData(),
    cuda_sum_of_hessians_buffer_.RawData(),
    num_used_indices,
    cuda_data_indices_in_leaf,
    cuda_hist_in_leaf,
    cuda_struct_.RawData());
  SynchronizeCUDADevice(__FILE__, __LINE__);
}

void CUDALeafSplits::LaunchInitValuesKernel(
  const double lambda_l1, const double lambda_l2,
  const data_size_t* cuda_bagging_data_indices,
  const data_size_t* cuda_data_indices_in_leaf,
  const data_size_t num_used_indices,
  hist_t* cuda_hist_in_leaf,
  const score_t* grad_scale,
  const score_t* hess_scale) {
  if (cuda_bagging_data_indices == nullptr) {
    CUDAInitValuesKernel3<false><<<num_blocks_init_from_gradients_, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
      reinterpret_cast<const int16_t*>(cuda_gradients_), num_used_indices, nullptr, cuda_sum_of_gradients_buffer_.RawData(),
      cuda_sum_of_hessians_buffer_.RawData(), cuda_sum_of_gradients_hessians_buffer_.RawData(), grad_scale, hess_scale);
  } else {
    CUDAInitValuesKernel3<true><<<num_blocks_init_from_gradients_, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
      reinterpret_cast<const int16_t*>(cuda_gradients_), num_used_indices, cuda_bagging_data_indices, cuda_sum_of_gradients_buffer_.RawData(),
      cuda_sum_of_hessians_buffer_.RawData(), cuda_sum_of_gradients_hessians_buffer_.RawData(), grad_scale, hess_scale);
  }

  SynchronizeCUDADevice(__FILE__, __LINE__);
  CUDAInitValuesKernel4<<<1, NUM_THREADS_PER_BLOCK_LEAF_SPLITS>>>(
    lambda_l1, lambda_l2,
    num_blocks_init_from_gradients_,
    cuda_sum_of_gradients_buffer_.RawData(),
    cuda_sum_of_hessians_buffer_.RawData(),
    cuda_sum_of_gradients_hessians_buffer_.RawData(),
    num_used_indices,
    cuda_data_indices_in_leaf,
    cuda_hist_in_leaf,
    cuda_struct_.RawData());
  SynchronizeCUDADevice(__FILE__, __LINE__);
}

}  // namespace LightGBM

#endif  // USE_CUDA
